#include "hip/hip_runtime.h"
/* Tim Zuijderduijn (s3620166) 2025
   main.cu
*/

#include "fineman.cu"
#include <hip/hip_runtime.h>

hipDevice_t cuDevice;
hipCtx_t cuContext;
hipModule_t cuModule;

void checkError(hipError_t error, std::string msg) {
  if (error != hipSuccess) {
    printf("%s: %d\n", msg.c_str(), error);
    exit(1);
  }
}

void initCuda(Digraph &G) {
  //initialize
  hipInit(0);
  checkError(hipDeviceGet(&cuDevice, 0), "cannot get device 0");
  checkError(hipCtxCreate(&cuContext, 0, cuDevice), "cannot create context");
}

int main(int argc, char** argv) {

  Digraph G;
  int startVertex = atoi(argv[1]);
  readGraph(G, argc, argv);

  printf("Number of vertices %d\n", G.numVertices);
  printf("Number of edges %d\n\n", G.numEdges);

  // vectors for results
  std::vector<int> distance(G.numVertices, std::numeric_limits<int>::max());
  std::vector<int> parent(G.numVertices, std::numeric_limits<int>::max());

  initCuda(G);

  // runs the bfs implementations
  // startBFS(G, startVertex, distance, parent);

  startFineman(G);

  return 0;

} // main