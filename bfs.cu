#include "hip/hip_runtime.h"
/* Tim Zuijderduijn (s3620166) 2025
   main.cc
*/

#include <cstdio>
#include <iostream>
#include <string>
#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>
#include <vector>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <thrust/unique.h>

#include "bfs_kernels.cu"
#include "digraph.h"

struct printf_functor
{
  __host__ __device__
  void operator()(int x)
  {
    // note that using printf in a __device__ function requires
    // code compiled for a GPU with compute capability 2.0 or
    // higher (nvcc --arch=sm_20)
    printf("%d ", x);
  }
};

// Used in augmented BFS to sort by vertex first and pivot ID second
struct sort_vertex_ID
{
  __host__ __device__
  bool operator()(thrust::tuple<int, int> const &a, thrust::tuple<int, int> const &b)
  {
    if (thrust::get<0>(a) < thrust::get<0>(b)) // vertex in a smaller than vertex in b
      return true;
    if (thrust::get<0>(a) > thrust::get<0>(b)) // vertex in a larger than vertex in b
      return false;
      
    return thrust::get<1>(a) < thrust::get<1>(b); // if vertices are equal, check for pivot ID
  }
};

void checkOutput(std::vector<int> &distance, std::vector<int> &expectedDistance, Digraph &G) {
  for (int i = 0; i < G.numVertices; i++) {
      if (distance[i] != expectedDistance[i]) {
          printf("%d %d %d\n", i, distance[i], expectedDistance[i]);
          printf("Wrong output!\n");
          exit(1);
      }
  }

  printf("Output OK!\n\n");
}

void initializeCudaBfs(int startVertex, std::vector<int> &distance, std::vector<int> &parent, Digraph &G,
                       thrust::device_vector<int> &d_distance,
                       thrust::device_vector<int> &d_parent,
                       thrust::device_vector<int> &d_currentQueue) {
  //initialize values
  std::fill(distance.begin(), distance.end(), std::numeric_limits<int>::max());
  std::fill(parent.begin(), parent.end(), std::numeric_limits<int>::max());
  distance[startVertex] = 0;
  parent[startVertex] = 0;

  d_distance = distance;
  d_parent = parent;

  thrust::fill_n(d_currentQueue.begin(), 1, startVertex);

}

void initializeCudaBfsAug(std::vector<int> &startVertices, Digraph &G,
                          thrust::device_vector<int> &d_currentQueue,
                          thrust::device_vector<int> &d_IDTagList,
                          thrust::device_vector<int> &d_queueID,
                          int IDTagSize) {

  // copy starting nodes to frontier
  thrust::copy(startVertices.begin(), startVertices.end(), d_currentQueue.begin());
  thrust::copy(startVertices.begin(), startVertices.end(), d_queueID.begin());

  // Init the ID tag list
  // G.numVertices * std::ceil(std::log(G.numVertices) entries in IDTagList

  int numEntries = G.numVertices * IDTagSize;
  std::vector<int> tempList(numEntries, -1);
  for (auto v : startVertices) {
    int temp = v * IDTagSize; 
    tempList[temp] = v;
  }

  d_IDTagList = tempList;

}

void runCudaBfs(int startVertex, Digraph &G, std::vector<int> &distance,
                std::vector<int> &parent, int numVertices,
                thrust::device_vector<int> &d_adjacencyList,
                thrust::device_vector<int> &d_edgesOffset,
                thrust::device_vector<int> &d_edgesSize,
                thrust::device_vector<int> &d_distance,
                thrust::device_vector<int> &d_parent,
                thrust::device_vector<int> &d_currentQueue,
                thrust::device_vector<int> &d_nextQueue,
                thrust::device_vector<int> &d_degrees) {
  
  initializeCudaBfs(startVertex, distance, parent, G,
                    d_distance, d_parent, d_currentQueue);

  //launch kernel
  printf("Starting standard parallel bfs.\n");
  auto start = std::chrono::steady_clock::now();

  int queueSize = 1;
  int nextQueueSize = 0;
  int level = 0;
  int maxLevel = std::ceil(std::pow(std::cbrt(numVertices), 2) * std::log(numVertices));
  bool reachedEnd = true;

  while (queueSize) {
      if (level >= maxLevel) {
        reachedEnd = false;
        break;
      }

      // next layer phase
      nextLayer<<<queueSize / 1024 + 1, 1024>>>
                (level,
                thrust::raw_pointer_cast(d_adjacencyList.data()),
                thrust::raw_pointer_cast(d_edgesOffset.data()),
                thrust::raw_pointer_cast(d_edgesSize.data()),
                thrust::raw_pointer_cast(d_distance.data()),
                thrust::raw_pointer_cast(d_parent.data()),
                queueSize,
                thrust::raw_pointer_cast(d_currentQueue.data()));
      // counting degrees phase
      countDegrees<<<queueSize / 1024 + 1, 1024>>>
                  (thrust::raw_pointer_cast(d_adjacencyList.data()),
                  thrust::raw_pointer_cast(d_edgesOffset.data()),
                  thrust::raw_pointer_cast(d_edgesSize.data()),
                  thrust::raw_pointer_cast(d_parent.data()),
                  queueSize,
                  thrust::raw_pointer_cast(d_currentQueue.data()),
                  thrust::raw_pointer_cast(d_degrees.data()));
      
      // doing scan on degrees
      thrust::inclusive_scan(d_degrees.begin(), d_degrees.begin() + queueSize, d_degrees.begin());
      nextQueueSize = d_degrees[queueSize - 1];

      // assigning vertices to nextQueue
      assignVerticesNextQueue<<<queueSize / 1024 + 1, 1024>>>
                              (thrust::raw_pointer_cast(d_adjacencyList.data()),
                               thrust::raw_pointer_cast(d_edgesOffset.data()),
                               thrust::raw_pointer_cast(d_edgesSize.data()),
                               thrust::raw_pointer_cast(d_parent.data()),
                               queueSize,
                               thrust::raw_pointer_cast(d_currentQueue.data()),
                               thrust::raw_pointer_cast(d_nextQueue.data()),
                               thrust::raw_pointer_cast(d_degrees.data()),
                               nextQueueSize);
      
      level++;
      queueSize = nextQueueSize;
      d_currentQueue.swap(d_nextQueue);
  }


  auto end = std::chrono::steady_clock::now();
  long duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
  printf("Elapsed time in milliseconds : %li ms.\n", duration);
  if (!reachedEnd) {
    printf("Did not reach end.\n");
  }
  
  std::cout << "\n" << level << " " << maxLevel << "\n";

}

void runCudaBfsAug(std::vector<int> startVertices, Digraph &G,
                   int distance, int numVertices, int IDTagSize,
                   thrust::device_vector<int> &d_adjacencyList,
                   thrust::device_vector<int> &d_edgesOffset,
                   thrust::device_vector<int> &d_edgesSize,
                   thrust::device_vector<int> &d_currentQueue,
                   thrust::device_vector<int> &d_nextQueue,
                   thrust::device_vector<int> &d_degrees,
                   thrust::device_vector<int> &d_IDTagList,
                   thrust::device_vector<int> &d_queueID,
                   thrust::device_vector<int> &d_nextQueueID) {

  initializeCudaBfsAug(startVertices, G, d_currentQueue,
                       d_IDTagList, d_queueID, IDTagSize);

  //launch kernel
  printf("Starting augmented parallel bfs.\n");
  auto start = std::chrono::steady_clock::now();

  int queueSize = startVertices.size();
  int nextQueueSize = 0;
  int level = 0;
  bool reachedEnd = true;

  // Used as a flag by assignPivotID()
  thrust::device_vector<bool> IDTagListOverflow(1, false);

  while (queueSize) {

      if (distance > -1 && level >= distance) {
        reachedEnd = false;
        break;
      }

      if (IDTagListOverflow[0]) {
        break;
      }

      // Counting degrees phase
      augCountDegrees<<<queueSize / 1024 + 1, 1024>>>
                      (thrust::raw_pointer_cast(d_adjacencyList.data()),
                      thrust::raw_pointer_cast(d_edgesOffset.data()),
                      thrust::raw_pointer_cast(d_edgesSize.data()),
                      queueSize,
                      thrust::raw_pointer_cast(d_currentQueue.data()),
                      thrust::raw_pointer_cast(d_degrees.data()));
      
      // Doing scan on degrees
      thrust::inclusive_scan(d_degrees.begin(), d_degrees.begin() + queueSize, d_degrees.begin());
      nextQueueSize = d_degrees[queueSize - 1];

      // Assigning vertices to nextQueue
      // Also checks the ID tag list
      augAssignVNQ<<<queueSize / 1024 + 1, 1024>>>
                    (thrust::raw_pointer_cast(d_adjacencyList.data()),
                    thrust::raw_pointer_cast(d_edgesOffset.data()),
                    thrust::raw_pointer_cast(d_edgesSize.data()),
                    queueSize,
                    thrust::raw_pointer_cast(d_currentQueue.data()),
                    thrust::raw_pointer_cast(d_nextQueue.data()),
                    thrust::raw_pointer_cast(d_degrees.data()),
                    nextQueueSize,
                    thrust::raw_pointer_cast(d_IDTagList.data()),
                    thrust::raw_pointer_cast(d_queueID.data()),
                    thrust::raw_pointer_cast(d_nextQueueID.data()),
                    IDTagSize);


      // Sorts values in d_nextQueue and d_nextQueueID
      // Sorts by vertex first, pivot ID second (so d_nextQueue first, d_nextQueueID second)
      auto iterSortFirst = thrust::make_zip_iterator(thrust::make_tuple(d_nextQueue.begin(), d_nextQueueID.begin()));
      auto iterSortLast = thrust::make_zip_iterator(thrust::make_tuple(d_nextQueue.begin() + nextQueueSize,
                                                                       d_nextQueueID.begin() + nextQueueSize));

      thrust::stable_sort(iterSortFirst, iterSortLast, sort_vertex_ID());

      // Compaction pass, duplication removal using d_nextQueue and d_nextQueueID
      auto iterUnique = thrust::unique(iterSortFirst, iterSortLast);
      if (iterUnique != iterSortLast) {
        thrust::fill(iterUnique, iterSortLast, thrust::make_tuple(0, -1));
        nextQueueSize = thrust::distance(iterUnique, iterSortLast);
      }

      // Assigns pivot IDS to d_IDTagList
      assignPivotID<<<nextQueueSize / 1024 + 1, 1024>>>
                    (thrust::raw_pointer_cast(d_nextQueue.data()),
                     thrust::raw_pointer_cast(d_nextQueueID.data()),
                     thrust::raw_pointer_cast(d_IDTagList.data()),
                     nextQueueSize,
                     IDTagSize,
                     thrust::raw_pointer_cast(IDTagListOverflow.data()));
      
      /*thrust::for_each(d_IDTagList.begin(), d_IDTagList.end(), printf_functor());
      std::cout << "\n";
      thrust::for_each(d_currentQueue.begin(), d_currentQueue.end(), printf_functor());
      std::cout << "\n";
      thrust::for_each(d_queueID.begin(), d_queueID.end(), printf_functor());
      std::cout << "\n";
      thrust::for_each(d_nextQueue.begin(), d_nextQueue.end(), printf_functor());
      std::cout << "\n";
      thrust::for_each(d_nextQueueID.begin(), d_nextQueueID.end(), printf_functor());
      std::cout << "\n" << nextQueueSize;
      std::cout << "\n\n";*/

      level++;
      queueSize = nextQueueSize;
      d_currentQueue.swap(d_nextQueue);
      d_queueID.swap(d_nextQueueID);
  }

  auto end = std::chrono::steady_clock::now();
  long duration = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();
  printf("Elapsed time in milliseconds : %li ms.\n", duration);

  if (!reachedEnd) {
    printf("Did not reach end.\n");
  }
  if (IDTagListOverflow[0]) {
    printf("ID Tag list has overflow.\n");
  }

}

void initDevVector(Digraph &G,
                   thrust::device_vector<int> &d_adjacencyList,
                   thrust::device_vector<int> &d_edgesOffset,
                   thrust::device_vector<int> &d_edgesSize,
                   thrust::device_vector<int> &d_distance,
                   thrust::device_vector<int> &d_parent,
                   thrust::device_vector<int> &d_currentQueue,
                   thrust::device_vector<int> &d_nextQueue,
                   thrust::device_vector<int> &d_degrees) {

  d_adjacencyList = G.adjacencyList;
  d_edgesOffset = G.edgesOffset;
  d_edgesSize = G.edgesSize;
  thrust::fill(d_distance.begin(), d_distance.end(), 0);
  thrust::fill(d_parent.begin(), d_parent.end(), 0);
  thrust::fill(d_currentQueue.begin(), d_currentQueue.end(), 0);
  thrust::fill(d_nextQueue.begin(), d_nextQueue.end(), 0);
  thrust::fill(d_degrees.begin(), d_degrees.end(), 0);

}

int startBFS(Digraph &G, int startVertex,
             std::vector<int> &distance, std::vector<int> &parent) {

  // device vectors for kernels
  thrust::device_vector<int> d_adjacencyList(G.adjacencyList);
  thrust::device_vector<int> d_edgesOffset(G.edgesOffset);
  thrust::device_vector<int> d_edgesSize(G.edgesSize);
  thrust::device_vector<int> d_distance(G.numVertices, 0);
  thrust::device_vector<int> d_parent(G.numVertices, 0);
  thrust::device_vector<int> d_currentQueue(G.numVertices, 0);
  thrust::device_vector<int> d_nextQueue(G.numVertices, 0);
  thrust::device_vector<int> d_degrees(G.numVertices, 0);

  // normal BFS
  runCudaBfs(startVertex, G, distance, parent, G.numVertices,
             d_adjacencyList, d_edgesOffset, d_edgesSize, d_distance,
             d_parent, d_currentQueue, d_nextQueue, d_degrees);


  int dD = -1;
  std::vector<int> startVertices = {0, 2};
  int IDTagSize = std::ceil(std::log(G.numVertices));
  thrust::device_vector<int> d_IDTagList(G.numVertices * IDTagSize);
  thrust::device_vector<int> d_queueID(G.numVertices, -1);
  thrust::device_vector<int> d_nextQueueID(G.numVertices, -1);

  initDevVector(G, d_adjacencyList, d_edgesOffset, d_edgesSize, d_distance,
                d_parent, d_currentQueue, d_nextQueue, d_degrees);

  // augmented BFS
  runCudaBfsAug(startVertices, G, dD, G.numVertices, IDTagSize,
                d_adjacencyList, d_edgesOffset, d_edgesSize,
                d_currentQueue, d_nextQueue, d_degrees,
                d_IDTagList, d_queueID, d_nextQueueID);

  return 0;

} // main
